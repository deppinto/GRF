#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "selfPropelledAligningParticleDynamics.cuh"

/** \file selfPropelledAligningParticleDynamics.cu
    * Defines kernel callers and kernels for GPU calculations of simple active 2D cell models
*/

/*!
    \addtogroup simpleEquationOfMotionKernels
    @{
*/

/*!
Each thread calculates the displacement of an individual cell
*/
__global__ void spp_aligning_eom_integration_kernel(double2 *forces,
                                           double2 *velocities,
                                           double2 *displacements,
                                           double2 *motility,
                                           double *cellDirectors,
                                           hiprandState *RNGs,
                                           int N,
                                           double deltaT,
                                           int Timestep,
                                           double mu,
                                           double J)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >=N)
        return;

    //get an appropriate random angle displacement
    hiprandState_t randState;
    randState=RNGs[idx];
    double v0 = motility[idx].x;
    double Dr = motility[idx].y;
    double angleDiff = cur_norm(&randState)*sqrt(2.0*deltaT*Dr);
    RNGs[idx] = randState;

    double currentTheta = cellDirectors[idx];
    if(currentTheta < -PI)
        currentTheta += 2*PI;
    if(currentTheta > PI)
        currentTheta -= 2*PI;
    //update displacements
    velocities[idx].x = v0*Cos(currentTheta) + mu*forces[idx].x;
    velocities[idx].y = v0*Sin(currentTheta) + mu*forces[idx].y;
    displacements[idx] = deltaT*velocities[idx];

    double currentPhi = atan2(displacements[idx].y,displacements[idx].x);

    //update director
    cellDirectors[idx] = currentTheta + angleDiff - deltaT*J*Sin(currentTheta-currentPhi);
    return;
    };

//!get the current timesteps vector of displacements into the displacement vector
bool gpu_spp_aligning_eom_integration(
                    double2 *forces,
                    double2 *velocities,
                    double2 *displacements,
                    double2 *motility,
                    double *cellDirectors,
                    hiprandState *RNGs,
                    int N,
                    double deltaT,
                    int Timestep,
                    double mu,
                    double J)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    spp_aligning_eom_integration_kernel<<<nblocks,block_size>>>(
                                forces,velocities,displacements,motility,cellDirectors,
                                RNGs,
                                N,deltaT,Timestep,mu, J);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
