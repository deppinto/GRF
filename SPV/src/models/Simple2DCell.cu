#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"
#include "Simple2DCell.cuh"

/** \file Simple2DCell.cu
    * Defines kernel callers and kernels for GPU calculations of simple 2D cell models
*/

/*!
    \addtogroup Simple2DCellKernels
    @{
*/

__host__ __device__ void moveDegreesOfFreedomFunction(int idx, double2 *d_points, double2 *d_disp, periodicBoundaries Box)
    {
    d_points[idx].x += d_disp[idx].x;
    d_points[idx].y += d_disp[idx].y;
    Box.putInBoxReal(d_points[idx]);
    return;
    };
__host__ __device__ void moveDegreesOfFreedomFunctionScaled(int idx, double2 *d_points, double2 *d_disp, double scale, periodicBoundaries Box)
    {
    d_points[idx].x += scale*d_disp[idx].x;
    d_points[idx].y += scale*d_disp[idx].y;
    Box.putInBoxReal(d_points[idx]);
    return;
    };

/*!
  A simple routine that takes in a pointer array of points, an array of displacements,
  adds the displacements to the points, and puts the points back in the primary unit cell.
*/
__global__ void gpu_move_degrees_of_freedom_kernel(double2 *d_points,
                                          double2 *d_disp,
                                          int N,
                                          periodicBoundaries Box
                                         )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    d_points[idx].x += d_disp[idx].x;
    d_points[idx].y += d_disp[idx].y;
    Box.putInBoxReal(d_points[idx]);
    return;
    };

/*!
  A simple routine that takes in a pointer array of points, an array of displacements,
  adds the displacements to the points, but with the displacement vector scaled by some amount, and
  puts the points back in the primary unit cell.
  This is useful, e.g., when the displacements are a dt times a velocity
*/
__global__ void gpu_move_degrees_of_freedom_kernel(double2 *d_points,
                                          double2 *d_disp,
                                          double scale,
                                          int N,
                                          periodicBoundaries Box
                                         )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    d_points[idx].x += scale*d_disp[idx].x;
    d_points[idx].y += scale*d_disp[idx].y;
    Box.putInBoxReal(d_points[idx]);
    return;
    };

/*!
  A simple routine that takes in a pointer array of points, an array of displacements,
  adds the displacements to the points, and puts the points back in the primary unit cell.
  Takes into account substrate interactions.
*/
__global__ void gpu_move_degrees_of_freedom_substrate_kernel(double2 *d_points,
                                          		     double2 *d_disp,
							     double2 *d_ap,
							     double *d_sg,
							     double *d_ct,
							     double dx,
							     double deltax,
							     double *d_tau,
                                          		     int N,
                                          		     periodicBoundaries Box
                                         		     )
    {
    // read in the particle that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    int x=d_points[idx].x/deltax;
    int y=d_points[idx].y/deltax;
    int site = x+y*dx;
    double tau=d_tau[idx];

    d_points[idx].x += d_disp[idx].x;
    d_points[idx].y += d_disp[idx].y;
    Box.putInBoxReal(d_points[idx]);

    x=d_points[idx].x/deltax;
    y=d_points[idx].y/deltax;
    int site2 = x+y*dx;
    double IncVal=d_sg[site2]-d_sg[site];
    d_ct[idx]=d_ct[idx]+IncVal;
    double NextVal=d_ct[idx];
    double Val=0;

    Val=d_ap[idx].y;
    d_ap[idx].y = NextVal+exp(-1/tau)*(Val-NextVal);

    return;
    };

/*!
every thread just writes in a value
*/
__global__ void gpu_set_integer_array_kernel(int *d_array,
                                          int value,
                                          int N)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    d_array[idx] = value;
    return;
    };

/*!
\param d_points double2 array of locations
\param d_disp   double2 array of displacements
\param N        The number of degrees of freedom to move
\param Box      The periodicBoundaries in which the new positions must reside
*/
bool gpu_move_degrees_of_freedom(double2 *d_points,
                        double2 *d_disp,
                        double  scale,
                        int N,
                        periodicBoundaries &Box
                        )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_move_degrees_of_freedom_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_disp,
                                                scale,
                                                N,
                                                Box
                                                );
    HANDLE_ERROR(hipGetLastError());

    return hipSuccess;
    };

/*move degrees of freedom substrate*/

bool gpu_move_degrees_of_freedom_substrate(double2 *d_points,
                        double2 *d_disp,
			double2 *d_ap,
			double *d_sg,
			double *d_ct,
                        double dx,
			double deltax,
                        double *d_tau,
                        int N,
                        periodicBoundaries &Box
                        )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_move_degrees_of_freedom_substrate_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_disp,
						d_ap,
						d_sg,
						d_ct,
						dx,
						deltax,
						d_tau,
                                                N,
                                                Box
                                                );
    HANDLE_ERROR(hipGetLastError());

    return hipSuccess;
    };

/*!
\param d_points double2 array of locations
\param d_disp   double2 array of displacements
\param N        The number of degrees of freedom to move
\param Box      The periodicBoundaries in which the new positions must reside
*/
bool gpu_move_degrees_of_freedom(double2 *d_points,
                        double2 *d_disp,
                        int N,
                        periodicBoundaries &Box
                        )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_move_degrees_of_freedom_kernel<<<nblocks,block_size>>>(
                                                d_points,
                                                d_disp,
                                                N,
                                                Box
                                                );
    HANDLE_ERROR(hipGetLastError());

    return hipSuccess;
    };

/*!
\param d_array int array of values
\param value   the integer to set the entire array to
\param N        The number of values in the array to set (d_array[0] tp d_array[N-1])
*/
bool gpu_set_integer_array(int *d_array,
                           int value,
                           int N
                          )
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;

    gpu_set_integer_array_kernel<<<nblocks,block_size>>>(
                                                d_array,
                                                value,
                                                N);
    HANDLE_ERROR(hipGetLastError());

    return hipSuccess;
    };

/** @} */ //end of group declaration
